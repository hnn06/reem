#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"
#define coarse_factor 2

__global__ void histogram_private_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {

    // TODO
    unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;
    

    
    __shared__ unsigned int private_histogram[NUM_BINS];
    if (threadIdx.x<NUM_BINS){
        private_histogram[threadIdx.x]=0;
    }
     __syncthreads();
    
    
    

        
    if(i < width*height) {
        unsigned char b = image[i];
        atomicAdd(&private_histogram[b], 1);
    }
    __syncthreads();
    if (threadIdx.x<NUM_BINS){
        atomicAdd(&bins[threadIdx.x],private_histogram[threadIdx.x]);
    }
    
    
    
}

void histogram_gpu_private(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

    // TODO
    unsigned int numThreadsPerBlock =1024;
    unsigned int numBlocks=(width*height+numThreadsPerBlock-1)/numThreadsPerBlock;
    histogram_private_kernel<<< numBlocks,numThreadsPerBlock>>> (image_d,bins_d,width,height);
    





}

__global__ void histogram_private_coarse_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {

    // TODO
    unsigned int i=coarse_factor*blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ unsigned int private_histogram[NUM_BINS];
    
    if (threadIdx.x<NUM_BINS){
        private_histogram[threadIdx.x]=0;
    }
    __syncthreads();
    
    for (unsigned int c=0;c<coarse_factor;++c){
        if(i+c*blockDim.x<width*height){
            unsigned char b = image[i+c*blockDim.x];
            atomicAdd(&private_histogram[b], 1);
        }
    }
    __syncthreads();
    if (threadIdx.x<NUM_BINS){
        atomicAdd(&bins[threadIdx.x],private_histogram[threadIdx.x]);
    }
 
    
              
   
    
        




}

void histogram_gpu_private_coarse(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

    // TODO
    unsigned int numThreadsPerBlock =1024;
    unsigned int numBlocks=(width*height+(numThreadsPerBlock*coarse_factor)-1)/(numThreadsPerBlock*coarse_factor);
    histogram_private_coarse_kernel<<< numBlocks,numThreadsPerBlock>>> (image_d,bins_d,width,height);
    
    





}

